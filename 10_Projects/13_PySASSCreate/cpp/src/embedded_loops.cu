#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <fstream>
#include <algorithm>
#include <format>
#include <vector>
#include <stdint.h>

/**
 * =======================================================================================================
 * This kernel is used as template that produces a lot of instructions using a lot of registers.
 * 
 * For custom cuda kernels, it is necessary to use a larger non-custom cuda kernel because the number of 
 * registers, that can be used, the amount of shared memory and just the space the compiled cuda kernel
 * takes inside the regular binary, are written into the binary.
 * 
 * Producing a custom cuda kernel from scratch will require writing all those values in the correct place
 * everywhere which then requires knowing exactly how to embed a cuda kernel inside a regular binary too.
 * 
 * These things are probably all documented somewhere, probably, partially inside of some people's heads.
 * =======================================================================================================
 * 
 * As opposed to sample_xdinstr_template.cu, this one also uses a 'double* d_output'. Using doubles introduces
 * a lot of floating point instructions into the kernel.
 * 
 * Using the 'output' pointer and reading from it with offsets into variables that are all seemingly
 * dependent on one another, produces a lot of registers. Since the compiler doesn't know what 'output'
 * is exactly, it can't optimize it away.
 * 
 * The for-loops without '#pragma unroll 1' just produce a lot of instructions, because the compiler will
 * unroll the loops a little bit, trading instruction space for less iterations.
 */

// __global__ void kernelT() {
__global__ void 
kernelT(unsigned int a, uint64_t *control, uint64_t *ui_output, double* d_output, uint64_t* ui_input, double* d_input) {
    if(a > 0x2710) return;

    // Leave this code be because it produces a lot of instructions that can be overwritten
    uint64_t b0 = static_cast<uint64_t>(a);
    uint64_t b1 = ui_output[0] + 1;
    uint64_t b2 = ui_output[1] + 1;
    uint64_t b3 = ui_output[2] + 1;

    uint64_t res4 = 0;
    for(uint64_t j=0; j<b2; ++j){
        int64_t t43 = clock64();
        int64_t t44 = clock64();
        res4 += 3UL*static_cast<uint64_t>(t44 - t43);
        if(b2 > 0x80 && j > 0x60) break;
    }
    d_output[0] = static_cast<double>(res4);

    uint64_t res5 = 0;
    for(uint64_t j=0; j<b2; ++j){
        int64_t t53 = clock64();
        int64_t t54 = clock64();
        res5 += static_cast<uint64_t>(t54 - t53);
        if(b2 > 0x90 && j > 0x70) return;
    }
    d_output[1] = static_cast<double>(res5);

    // outer loop ========>
    int64_t t0 = clock64();
    #pragma unroll 1
    for(uint64_t i=0; i<b0; ++i){
        // nested loop 1 ========>
        uint64_t res1 = 0;
        int64_t t11 = clock64();
        #pragma unroll 1
        for(uint64_t j=0; j<b1; ++j){
            int64_t t13 = clock64();
            // nested-nested loop 3 ========>
            uint64_t res3 = 0;
            int64_t t31 = clock64();
            #pragma unroll 1
            for(uint64_t x=0; x<b3; ++x){
                int64_t t33 = clock64();
                int64_t t34 = clock64();
                res3 += static_cast<uint64_t>(t34 - t33);
            }
            int64_t t32 = clock64();
            ui_output[i+3] = res3 + static_cast<uint64_t>((static_cast<float>(t32-t31) + 1.358f));
            // ========> nested-nested loop 3

            int64_t t14 = clock64();
            res1 += static_cast<uint64_t>(t14 - t13);
        }
        int64_t t12 = clock64();
        ui_output[i+1] = res1 + static_cast<uint64_t>((static_cast<float>(t12-t11) + 1.256f));
        // ========> nested loop 1

        if(a < 10) return;

        // nested loop 2 ========>
        uint64_t b2 = res1 + 1;
        uint64_t res2 = 0;
        int64_t t21 = clock64();
        #pragma unroll 1
        for(uint64_t j=0; j<b2; ++j){
            int64_t t23 = clock64();
            int64_t t24 = clock64();
            res2 += static_cast<uint64_t>(t24 - t24);
        }
        int64_t t22 = clock64();
        ui_output[i+2] = res2 + static_cast<uint64_t>((static_cast<float>(t22-t21) + 1.2869f));
        // ========> nested loop 2
    }
    int64_t t3 = clock64();
    ui_output[0] = static_cast<int>(a * (static_cast<float>(t3-t0) + 1.28964f*static_cast<float>(a)));
    // ========> outer loop

    return;
}

int main(int argc, char** argv){
    std::string fn = std::string(argv[0]);
    int split_ind = fn.find_last_of("/\\") + 1;
    std::string path = fn.substr(0,split_ind);
    std::string filename = fn.substr(split_ind);

    if(argc != 2){
        std::cout << std::vformat("{0} [input]", std::make_format_args(fn)) << std::endl;
        return 0;
    }
    unsigned int input = static_cast<unsigned int>(std::stoi(argv[1]));
    std::cout << "[Filename]" << filename << "[/Filename] [Input]" << input << "[/Input]" << std::endl;

    int loop_count = (input == 0) ? input++ : input;

    for(int lp=0; lp<loop_count; ++lp){
        std::cout << "[LoopCount_" << lp << "]" << std::endl;
        // ==================================================================================================
        // Define input data
        // ==================================================================================================
        int control_size = 15;
        int printout = loop_count;
        std::vector<uint64_t> control(control_size);
        std::vector<uint64_t> ui_output(printout);
        std::vector<double> d_output(printout);
        std::vector<uint64_t> ui_input(printout);
        std::vector<double> d_input(printout);
        for(int i=0; i<control.size(); ++i) control[i] = i+100;
        for(int i=0; i<ui_output.size(); ++i) ui_output[i] = i+1001;
        for(int i=0; i<d_output.size(); ++i) d_output[i] = static_cast<double>(i)+2001.2;
        for(int i=0; i<ui_input.size(); ++i) ui_input[i] = 0;
        for(int i=0; i<d_input.size(); ++i) d_input[i] = 0.0;

        // ==================================================================================================
        // Print input data
        // ==================================================================================================
        std::cout << "[BeforeKernel]" << std::endl;
        std::cout << " + [C] Before kernel " << filename << std::endl << "       finished: control(i)[";
        for(int i=0; i<control_size; ++i) {
            std::cout << i;
            if(i<(control_size-1)) std::cout << ", ";
        }
        std::cout << "] = [Control]" << std::endl << "                   ";
        for(int i=0; i<control_size; ++i){
            std::cout << control[i];
            if(i<(control_size-1)) std::cout << ", ";
        }
        std::cout << "[/Control]" << std::endl;

        std::cout << " + [IO] Before kernel " << filename << std::endl << "       finished: ui_output(i)[";
        for(int i=0; i<printout; ++i) {
            std::cout << i;
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "] = [UiOutput]" << std::endl << "                   ";
        for(int i=0; i<printout; ++i){
            std::cout << ui_output[i];
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "[/UiOutput]" << std::endl;

        std::cout << " + [DO] Before kernel " << filename << std::endl << "       finished: d_output(d)[";
        for(int i=0; i<printout; ++i) {
            std::cout << i;
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "] = [DOutput]" << std::endl << "                   ";
        for(int i=0; i<printout; ++i){
            std::cout << d_output[i];
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "[/DOutput]" << std::endl;

        std::cout << " + [II] Before kernel " << filename << std::endl << "       finished: ui_input(i)[";
        for(int i=0; i<printout; ++i) {
            std::cout << i;
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "] = [UiInput]" << std::endl << "                   ";
        for(int i=0; i<printout; ++i){
            std::cout << ui_input[i];
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "[/UiInput]" << std::endl;

        std::cout << " + [DI] Before kernel " << filename << std::endl << "       finished: d_input(d)[";
        for(int i=0; i<printout; ++i) {
            std::cout << i;
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "] = [DInput]" << std::endl << "                   ";
        for(int i=0; i<printout; ++i){
            std::cout << d_input[i];
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "[/DInput]" << std::endl;

        std::cout <<" + [.][Before kernel] [UiInput0]" << ui_input[0] << "[/UiInput0], [DInput0]" << d_input[0] << "[/DInput0]" << std::endl;

        // ==================================================================================================
        // Copy input data to GPU
        // ==================================================================================================
        uint64_t* device_control;
        hipMalloc(&device_control, control.size()*sizeof(uint64_t));
        hipMemcpy(device_control, control.data(), control.size()*sizeof(uint64_t), hipMemcpyHostToDevice);

        uint64_t* device_ui_output;
        hipMalloc(&device_ui_output, ui_output.size()*sizeof(uint64_t));
        hipMemcpy(device_ui_output, ui_output.data(), ui_output.size()*sizeof(uint64_t), hipMemcpyHostToDevice);

        double* device_d_output;
        hipMalloc(&device_d_output, d_output.size()*sizeof(double));
        hipMemcpy(device_d_output, d_output.data(), d_output.size()*sizeof(double), hipMemcpyHostToDevice);

        uint64_t* device_ui_input;
        hipMalloc(&device_ui_input, ui_input.size()*sizeof(uint64_t));
        hipMemcpy(device_ui_input, ui_input.data(), ui_input.size()*sizeof(uint64_t), hipMemcpyHostToDevice);

        double* device_d_input;
        hipMalloc(&device_d_input, d_input.size()*sizeof(double));
        hipMemcpy(device_d_input, d_input.data(), d_input.size()*sizeof(double), hipMemcpyHostToDevice);

        std::cout << "[/BeforeKernel]" << std::endl;
        // ==================================================================================================
        // Run Kernel
        // ==================================================================================================
        kernelT<<<1,1>>>(input, device_control, device_ui_output, device_d_output, device_ui_input, device_d_input);

        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if ( err != hipSuccess ) {
            std::cout << "[CUDAError]" << hipGetErrorString(err) << "[/CUDAError]" << std::endl;
        }

        std::cout << "[AfterKernel]" << std::endl;

        // ==================================================================================================
        // Copy output data to CPU
        // ==================================================================================================
        hipMemcpy(control.data(), device_control, control.size()*sizeof(uint64_t), hipMemcpyDeviceToHost);
        hipMemcpy(ui_output.data(), device_ui_output, ui_output.size()*sizeof(uint64_t), hipMemcpyDeviceToHost);
        hipMemcpy(d_output.data(), device_d_output, d_output.size()*sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(ui_input.data(), device_ui_input, ui_input.size()*sizeof(uint64_t), hipMemcpyDeviceToHost);
        hipMemcpy(d_input.data(), device_d_input, d_input.size()*sizeof(double), hipMemcpyDeviceToHost);
        hipFree(device_control);
        hipFree(device_ui_output);
        hipFree(device_d_output);
        hipFree(device_ui_input);
        hipFree(device_d_input);

        // ==================================================================================================
        // Print output data
        // ==================================================================================================
        std::cout << " + [C] After kernel " << filename << std::endl << "       finished: control(i)[";
        for(int i=0; i<control_size; ++i) {
            std::cout << i;
            if(i<(control_size-1)) std::cout << ", ";
        }
        std::cout << "] = [Control]" << std::endl << "                   ";
        for(int i=0; i<control_size; ++i){
            std::cout << control[i];
            if(i<(control_size-1)) std::cout << ", ";
        }
        std::cout << "[/Control]" << std::endl;

        std::cout << " + [I] After kernel " << filename << std::endl << "       finished: ui_output(i)[";
        for(int i=0; i<printout; ++i) {
            std::cout << i;
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "] = [UiOutput]" << std::endl << "                   ";
        for(int i=0; i<printout; ++i){
            std::cout << ui_output[i];
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "[/UiOutput]" << std::endl;

        std::cout << " + [D] After kernel " << filename << std::endl << "       finished: d_output(d)[";
        for(int i=0; i<printout; ++i) {
            std::cout << i;
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "] = [DOutput]" << std::endl << "                   ";
        for(int i=0; i<printout; ++i){
            std::cout << d_output[i];
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "[/DOutput]" << std::endl;

        std::cout << " + [II] After kernel " << filename << std::endl << "       finished: ui_input(i)[";
        for(int i=0; i<printout; ++i) {
            std::cout << i;
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "] = [UiInput]" << std::endl << "                   ";
        for(int i=0; i<printout; ++i){
            std::cout << ui_input[i];
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "[/UiInput]" << std::endl;

        std::cout << " + [DI] After kernel " << filename << std::endl << "       finished: d_input(d)[";
        for(int i=0; i<printout; ++i) {
            std::cout << i;
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "] = [DInput]" << std::endl << "                   ";
        for(int i=0; i<printout; ++i){
            std::cout << d_input[i];
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "[/DInput]" << std::endl;

        std::cout <<" + [.][After kernel] [UiInput0]" << ui_input[0] << "[/UiInput0], [DInput0]" << d_input[0] << "[/DInput0]" << std::endl;
        std::cout << "[/AfterKernel]" << std::endl;

        std::cout << "[/LoopCount_" << lp << "]" << std::endl;
    }

    return 0;
}