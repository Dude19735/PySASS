#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <fstream>
#include <algorithm>
#include <format>
#include <vector>
#include <stdint.h>

/**
 * ===========================================================================================================
 * This kernel can be used as template that produces around 60 (SM 70 to 86) to 75 (SM 90 to 120) instructions
 * ===========================================================================================================
 * Number of instructions can be modulated a little bit by setting
 *  - #pragma unroll 1 => fewest instructions
 *  - #pragma unroll 2 => around 60 to 75 instructions
 *  -  ...             => more than 60 to 75 instructions
 * 
 * NOTE: check out the template generator for the stuff that is actually being used!
 */

// __global__ void kernelT() {
__global__ void 
kernelT(unsigned int a, uint64_t *control, uint64_t *ui_output, double* d_output, uint64_t* ui_input, double* d_input, uint64_t *clk_out_1, float* f_output) {
    // #pragma unroll 1
    // for(unsigned int i=0; i<a; ++i){
    //     int64_t t1 = clock64();
    //     int64_t t2 = clock64();
    //     f_output[i] = static_cast<float>(a * (static_cast<float>(t2-t1) + 1.256f));
    // }
    ui_output[0] = ui_input[0] / a;
    // d_output[0] = static_cast<double>(ui_input[0]) / static_cast<double>(a);
    return;
}

int run_test(void* kernel_ptr, int input, int loop_count, const std::string filename, const std::string enc_vals){
    for(int lp=0; lp<loop_count; ++lp){
        std::cout << "[LoopCount_" << lp << "]" << std::endl;
        // ==================================================================================================
        // Define input data
        // ==================================================================================================
        int control_size = 15;
        int printout = loop_count;
        std::vector<uint64_t> control(control_size);
        std::vector<uint64_t> ui_output(printout);
        std::vector<double> d_output(printout);
        std::vector<float> f_output(printout);
        std::vector<uint64_t> ui_input(printout);
        std::vector<double> d_input(printout);
        std::vector<uint64_t> clk_out_1(printout);
        for(int i=0; i<control.size(); ++i) control[i] = i+100;
        for(int i=0; i<ui_output.size(); ++i) ui_output[i] = i+1001;
        for(int i=0; i<d_output.size(); ++i) d_output[i] = static_cast<double>(i)+2001.2;
        for(int i=0; i<f_output.size(); ++i) f_output[i] = static_cast<float>(i)+3001.3;
        for(int i=0; i<ui_input.size(); ++i) ui_input[i] = 0;
        for(int i=0; i<d_input.size(); ++i) d_input[i] = 0.0;
        for(int i=0; i<clk_out_1.size(); ++i) clk_out_1[i] = 999999998;

        // print enc vals
        std::cout << "[EncVals]" << std::endl;
        std::cout << enc_vals << std::endl;
        std::cout << "[/EncVals]" << std::endl;

        // ==================================================================================================
        // Print input data
        // ==================================================================================================
        std::cout << "[BeforeKernel]" << std::endl;
        std::cout << " + [C] Before kernel " << filename << std::endl << "       finished: control(i)[";
        for(int i=0; i<control_size; ++i) {
            std::cout << i;
            if(i<(control_size-1)) std::cout << ", ";
        }
        std::cout << "] = [Control]" << std::endl << "                   ";
        for(int i=0; i<control_size; ++i){
            std::cout << control[i];
            if(i<(control_size-1)) std::cout << ", ";
        }
        std::cout << "[/Control]" << std::endl;

        std::cout << " + [IO] Before kernel " << filename << std::endl << "       finished: ui_output(i)[";
        for(int i=0; i<printout; ++i) {
            std::cout << i;
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "] = [UiOutput]" << std::endl << "                   ";
        for(int i=0; i<printout; ++i){
            std::cout << ui_output[i];
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "[/UiOutput]" << std::endl;

        std::cout << " + [DO] Before kernel " << filename << std::endl << "       finished: d_output(d)[";
        for(int i=0; i<printout; ++i) {
            std::cout << i;
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "] = [DOutput]" << std::endl << "                   ";
        for(int i=0; i<printout; ++i){
            std::cout << d_output[i];
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "[/DOutput]" << std::endl;

        std::cout << " + [FO] Before kernel " << filename << std::endl << "       finished: f_output(f)[";
        for(int i=0; i<printout; ++i) {
            std::cout << i;
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "] = [FOutput]" << std::endl << "                   ";
        for(int i=0; i<printout; ++i){
            std::cout << f_output[i];
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "[/FOutput]" << std::endl;

        std::cout << " + [II] Before kernel " << filename << std::endl << "       finished: ui_input(i)[";
        for(int i=0; i<printout; ++i) {
            std::cout << i;
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "] = [UiInput]" << std::endl << "                   ";
        for(int i=0; i<printout; ++i){
            std::cout << ui_input[i];
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "[/UiInput]" << std::endl;

        std::cout << " + [DI] Before kernel " << filename << std::endl << "       finished: d_input(d)[";
        for(int i=0; i<printout; ++i) {
            std::cout << i;
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "] = [DInput]" << std::endl << "                   ";
        for(int i=0; i<printout; ++i){
            std::cout << d_input[i];
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "[/DInput]" << std::endl;

        std::cout << " + [C1] Before kernel " << filename << std::endl << "       finished: clk_out_1(i)[";
        for(int i=0; i<printout; ++i) {
            std::cout << i;
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "] = [ClkOut1]" << std::endl << "                   ";
        for(int i=0; i<printout; ++i){
            std::cout << clk_out_1[i];
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "[/ClkOut1]" << std::endl;

        std::cout <<" + [.][Before kernel] [UiInput0]" << ui_input[0] << "[/UiInput0], [DInput0]" << d_input[0] << "[/DInput0]" << std::endl;

        // ==================================================================================================
        // Copy input data to GPU
        // ==================================================================================================
        uint64_t* device_control;
        hipMalloc(&device_control, control.size()*sizeof(uint64_t));
        hipMemcpy(device_control, control.data(), control.size()*sizeof(uint64_t), hipMemcpyHostToDevice);

        uint64_t* device_ui_output;
        hipMalloc(&device_ui_output, ui_output.size()*sizeof(uint64_t));
        hipMemcpy(device_ui_output, ui_output.data(), ui_output.size()*sizeof(uint64_t), hipMemcpyHostToDevice);

        double* device_d_output;
        hipMalloc(&device_d_output, d_output.size()*sizeof(double));
        hipMemcpy(device_d_output, d_output.data(), d_output.size()*sizeof(double), hipMemcpyHostToDevice);

        float* device_f_output;
        hipMalloc(&device_f_output, f_output.size()*sizeof(float));
        hipMemcpy(device_f_output, f_output.data(), f_output.size()*sizeof(float), hipMemcpyHostToDevice);

        uint64_t* device_ui_input;
        hipMalloc(&device_ui_input, ui_input.size()*sizeof(uint64_t));
        hipMemcpy(device_ui_input, ui_input.data(), ui_input.size()*sizeof(uint64_t), hipMemcpyHostToDevice);

        double* device_d_input;
        hipMalloc(&device_d_input, d_input.size()*sizeof(double));
        hipMemcpy(device_d_input, d_input.data(), d_input.size()*sizeof(double), hipMemcpyHostToDevice);

        uint64_t* device_clk_out_1;
        hipMalloc(&device_clk_out_1, clk_out_1.size()*sizeof(uint64_t));
        hipMemcpy(device_clk_out_1, clk_out_1.data(), clk_out_1.size()*sizeof(uint64_t), hipMemcpyHostToDevice);

        std::cout << "[/BeforeKernel]" << std::endl;
        // ==================================================================================================
        // Run Kernel
        // ==================================================================================================
        // kernelT<<<1,1>>>(input, device_control, device_ui_output, device_d_output, device_ui_input, device_d_input, device_clk_out_1, device_f_output);
        void* args[] = {&input, &device_control, &device_ui_output, &device_d_output, &device_ui_input, &device_d_input, &device_clk_out_1, &device_f_output};
        hipLaunchKernel(reinterpret_cast<const void*>(kernel_ptr), 1, 1, args, 0, nullptr);

        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if ( err != hipSuccess ) {
            std::cout << "[CUDAError]" << hipGetErrorString(err) << "[/CUDAError]" << std::endl;
        }

        std::cout << "[AfterKernel]" << std::endl;

        // ==================================================================================================
        // Copy output data to CPU
        // ==================================================================================================
        hipMemcpy(control.data(), device_control, control.size()*sizeof(uint64_t), hipMemcpyDeviceToHost);
        hipMemcpy(ui_output.data(), device_ui_output, ui_output.size()*sizeof(uint64_t), hipMemcpyDeviceToHost);
        hipMemcpy(d_output.data(), device_d_output, d_output.size()*sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(f_output.data(), device_f_output, f_output.size()*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(ui_input.data(), device_ui_input, ui_input.size()*sizeof(uint64_t), hipMemcpyDeviceToHost);
        hipMemcpy(d_input.data(), device_d_input, d_input.size()*sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(clk_out_1.data(), device_clk_out_1, clk_out_1.size()*sizeof(uint64_t), hipMemcpyDeviceToHost);
        hipFree(device_control);
        hipFree(device_ui_output);
        hipFree(device_d_output);
        hipFree(device_f_output);
        hipFree(device_ui_input);
        hipFree(device_d_input);
        hipFree(device_clk_out_1);

        // ==================================================================================================
        // Print output data
        // ==================================================================================================
        std::cout << " + [C] After kernel " << filename << std::endl << "       finished: control(i)[";
        for(int i=0; i<control_size; ++i) {
            std::cout << i;
            if(i<(control_size-1)) std::cout << ", ";
        }
        std::cout << "] = [Control]" << std::endl << "                   ";
        for(int i=0; i<control_size; ++i){
            std::cout << control[i];
            if(i<(control_size-1)) std::cout << ", ";
        }
        std::cout << "[/Control]" << std::endl;

        std::cout << " + [I] After kernel " << filename << std::endl << "       finished: ui_output(i)[";
        for(int i=0; i<printout; ++i) {
            std::cout << i;
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "] = [UiOutput]" << std::endl << "                   ";
        for(int i=0; i<printout; ++i){
            std::cout << ui_output[i];
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "[/UiOutput]" << std::endl;

        std::cout << " + [D] After kernel " << filename << std::endl << "       finished: d_output(d)[";
        for(int i=0; i<printout; ++i) {
            std::cout << i;
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "] = [DOutput]" << std::endl << "                   ";
        for(int i=0; i<printout; ++i){
            std::cout << d_output[i];
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "[/DOutput]" << std::endl;

        std::cout << " + [F] After kernel " << filename << std::endl << "       finished: f_output(f)[";
        for(int i=0; i<printout; ++i) {
            std::cout << i;
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "] = [FOutput]" << std::endl << "                   ";
        for(int i=0; i<printout; ++i){
            std::cout << f_output[i];
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "[/FOutput]" << std::endl;

        std::cout << " + [II] After kernel " << filename << std::endl << "       finished: ui_input(i)[";
        for(int i=0; i<printout; ++i) {
            std::cout << i;
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "] = [UiInput]" << std::endl << "                   ";
        for(int i=0; i<printout; ++i){
            std::cout << ui_input[i];
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "[/UiInput]" << std::endl;

        std::cout << " + [DI] After kernel " << filename << std::endl << "       finished: d_input(d)[";
        for(int i=0; i<printout; ++i) {
            std::cout << i;
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "] = [DInput]" << std::endl << "                   ";
        for(int i=0; i<printout; ++i){
            std::cout << d_input[i];
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "[/DInput]" << std::endl;

        std::cout << " + [C1] After kernel " << filename << std::endl << "       finished: clk_out_1(i)[";
        for(int i=0; i<printout; ++i) {
            std::cout << i;
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "] = [ClkOut1]" << std::endl << "                   ";
        for(int i=0; i<printout; ++i){
            std::cout << clk_out_1[i];
            if(i<(printout-1)) std::cout << ", ";
        }
        std::cout << "[/ClkOut1]" << std::endl;

        std::cout <<" + [.][After kernel] [UiInput0]" << ui_input[0] << "[/UiInput0], [DInput0]" << d_input[0] << "[/DInput0]" << std::endl;
        std::cout << "[/AfterKernel]" << std::endl;

        std::cout << "[/LoopCount_" << lp << "]" << std::endl;
    }

    return 0;
}

int main(int argc, char** argv){
    std::string fn = std::string(argv[0]);
    int split_ind = fn.find_last_of("/\\") + 1;
    std::string path = fn.substr(0,split_ind);
    std::string filename = fn.substr(split_ind);

    if(argc != 2){
        std::cout << std::vformat("{0} [input]", std::make_format_args(fn)) << std::endl;
        return 0;
    }
    unsigned int input = static_cast<unsigned int>(std::stoi(argv[1]));
    std::cout << "[Filename]" << filename << "[/Filename] [Input]" << input << "[/Input]" << std::endl;

    int loop_count = (input == 0) ? input++ : input;

    run_test(reinterpret_cast<void*>(&kernelT), input, loop_count, filename, "");

    return 0;
}