#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <fstream>
#include <algorithm>
#include <format>
#include <vector>
#include <stdint.h>
#include <stdio.h>

/**
 * ===========================================================================================================
 * This kernel has two integer parameters
 * ===========================================================================================================
 * Use it for the tutorial that uses two integer arguments to showcase alignment
 */

// __global__ void kernelT() {
__global__ void 
kernelT1(unsigned int a, unsigned int b) {
    printf("param a=%u, param b=%u\n", a, b);
    return;
}

__global__ void 
kernelT2(unsigned int a, unsigned int* b) {
    printf("param a=%u, param b=%u\n", a, *b);
    return;
}

__global__ void 
kernelT3(unsigned int* a, unsigned int b, unsigned int c) {
    printf("param a=%u, param b=%u, param c=%u\n", *a, b, c);
    return;
}

int main(int argc, char** argv){
    std::string fn = std::string(argv[0]);
    int split_ind = fn.find_last_of("/\\") + 1;
    std::string path = fn.substr(0,split_ind);
    std::string filename = fn.substr(split_ind);

    if(argc != 3){
        std::cout << std::vformat("{0} [a] [b]", std::make_format_args(fn)) << std::endl;
        return 0;
    }
    unsigned int a = static_cast<unsigned int>(std::stoi(argv[1]));
    unsigned int b = static_cast<unsigned int>(std::stoi(argv[2]));

    kernelT1<<<1,1>>>(a, b);
    hipDeviceSynchronize();
    kernelT2<<<1,1>>>(a, &b);
    hipDeviceSynchronize();
    kernelT3<<<1,1>>>(&a, b, 3);
    hipDeviceSynchronize();

    return 0;
}